#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <map>
#include <random>
#include <tuple>
#include <utility>
#include <vector>

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

__device__ inline void cp_async4(void *smem_ptr, const void *glob_ptr) {
    const int BYTES = 16;
    uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    asm volatile(
        "{\n"
        "   cp.async.cg.shared.global [%0], [%1], %2;\n"
        "}\n" ::"r"(smem),
        "l"(glob_ptr),
        "n"(BYTES));
}

__device__ __forceinline__ void async_memcpy_waitall() {
    asm volatile("cp.async.wait_all;\n" ::);
}

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Too slow to actually run!)
//
// void matmul_cpu_naive(
//     int32_t size_i,
//     int32_t size_j,
//     int32_t size_k,
//     float const *a,
//     float const *b,
//     float *c) {
//     for (int32_t i = 0; i < size_i; ++i) {
//         for (int32_t j = 0; j < size_j; ++j) {
//             float sum = 0.0;
//             for (int32_t k = 0; k < size_k; ++k) {
//                 sum += a[i * size_k + k] * b[k * size_j + j];
//             }
//             c[i * size_j + j] = sum;
//         }
//     }
// }

/// <--- your code here --->

/*
    // OPTIONAL: Uncomment this block to include your kernel implementation
    // from Lab 4 for easy comparison.

    ////////////////////////////////////////////////////////////////////////////////
    // GPU Implementation with Reuse in L1/Shmem and Registers (Baseline from Lab 4)

    #define HAS_LAB_4_BASELINE_IMPL // <~~ keep this line if you want to benchmark your Lab 4 kernel!

    namespace matmul_l1_reg {

    __global__ void matmul_l1_reg(
        int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c) {
        // TODO: your GPU code here
    }

    void launch_matmul_l1_reg(
        int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c) {
        // TODO: your CPU code here
    }

    } // namespace matmul_l1_reg
*/

////////////////////////////////////////////////////////////////////////////////
// Optimized GPU Implementation

namespace matmul_improved {

__global__ void matmul_improved(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a, /* pointer to GPU memory */
    float const *b, /* pointer to GPU memory */
    float *c /* pointer to GPU memory */) {
    /* TODO: your GPU code here */
}

void launch_matmul_improved(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a, /* pointer to GPU memory */
    float const *b, /* pointer to GPU memory */
    float *c /* pointer to GPU memory */) {
    /* TODO: your CPU code here */
}

}; // namespace matmul_improved

////////////////////////////////////////////////////////////////////////////////
// Optimized GPU Implementation with Reduction along k

namespace matmul_improved_reduce {

/* TODO: your GPU kernels here... */

size_t get_workspace_size(int32_t size_i, int32_t size_j, int32_t size_k) {
    /* TODO: your CPU code here */
    return 0;
}

void launch_matmul_improved_reduce(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a, /* pointer to GPU memory */
    float const *b, /* pointer to GPU memory */
    float *c,       /* pointer to GPU memory */
    void *workspace /* pointer to GPU memory */
) {
    /* TODO: your CPU code here */
}

}; // namespace matmul_improved_reduce

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

std::vector<float> read_data(std::string const &path, int32_t size) {
    std::ifstream file(path, std::ios::binary);
    std::vector<float> data(size);
    file.read(reinterpret_cast<char *>(data.data()), data.size() * sizeof(float));
    if (file.fail()) {
        std::cerr << "Failed to read " << path << std::endl;
        std::abort();
    }
    return data;
}

template <typename Reset, typename F>
double
benchmark_ms(double target_time_ms, int32_t num_iters_inner, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t i = 0; i < num_iters_inner; ++i) {
            f();
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms / num_iters_inner);
    }
    return best_time_ms;
}

struct BenchmarkConfig {
    int32_t size_i;
    int32_t size_j;
    int32_t size_k;
};

struct TestData {
    std::map<std::tuple<int32_t, int32_t>, std::vector<float>> a;
    std::map<std::tuple<int32_t, int32_t>, std::vector<float>> b;
    std::map<std::tuple<int32_t, int32_t, int32_t>, std::vector<float>> c;
};

TestData read_test_data(
    std::string const &test_data_dir,
    std::vector<BenchmarkConfig> const &configs) {
    auto data = TestData{};
    for (auto const &config : configs) {
        auto size_i = config.size_i;
        auto size_j = config.size_j;
        auto size_k = config.size_k;

        auto path_prefix = test_data_dir + "/test_";

        if (data.a.find({size_i, size_k}) == data.a.end()) {
            data.a[{size_i, size_k}] = read_data(
                path_prefix + "a_" + std::to_string(size_i) + "x" +
                    std::to_string(size_k) + ".bin",
                size_i * size_k);
        }

        if (data.b.find({size_k, size_j}) == data.b.end()) {
            data.b[{size_k, size_j}] = read_data(
                path_prefix + "b_" + std::to_string(size_k) + "x" +
                    std::to_string(size_j) + ".bin",
                size_k * size_j);
        }

        if (data.c.find({size_i, size_j, size_k}) == data.c.end()) {
            data.c[{size_i, size_j, size_k}] = read_data(
                path_prefix + "c_" + std::to_string(size_i) + "x" +
                    std::to_string(size_j) + "x" + std::to_string(size_k) + ".bin",
                size_i * size_j);
        }
    }
    return data;
}

struct BenchmarkResults {
    char const *name;
    std::map<std::tuple<int32_t, int32_t, int32_t>, double> elapsed_ms;
};

enum class Phase {
    WARMUP,
    BENCHMARK,
};

template <typename Impl>
void run_config(
    Phase phase,
    TestData const &data,
    BenchmarkConfig const &config,
    BenchmarkResults &results) {
    auto size_i = config.size_i;
    auto size_j = config.size_j;
    auto size_k = config.size_k;

    auto const &a = data.a.at({size_i, size_k});
    auto const &b = data.b.at({size_k, size_j});
    auto const &c = data.c.at({size_i, size_j, size_k});

    float *a_gpu;
    float *b_gpu;
    float *c_gpu;
    CUDA_CHECK(hipMalloc(&a_gpu, size_i * size_k * sizeof(float)));
    CUDA_CHECK(hipMalloc(&b_gpu, size_k * size_j * sizeof(float)));
    CUDA_CHECK(hipMalloc(&c_gpu, size_i * size_j * sizeof(float)));

    CUDA_CHECK(hipMemcpy(
        a_gpu,
        a.data(),
        size_i * size_k * sizeof(float),
        hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(
        b_gpu,
        b.data(),
        size_k * size_j * sizeof(float),
        hipMemcpyHostToDevice));

    size_t workspace_size = Impl::get_workspace_size(size_i, size_j, size_k);
    void *workspace_gpu = nullptr;
    if (workspace_size > 0) {
        CUDA_CHECK(hipMalloc(&workspace_gpu, workspace_size));
        CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));
    }

    if (phase == Phase::BENCHMARK) {
        printf("  %6d  %6d  %6d", size_i, size_j, size_k);
    } else {
        printf("  warmup %6d  %6d  %6d", size_i, size_j, size_k);
    }

    Impl::run(size_i, size_j, size_k, a_gpu, b_gpu, c_gpu, workspace_gpu);

    std::vector<float> c_out_host(size_i * size_j);
    CUDA_CHECK(hipMemcpy(
        c_out_host.data(),
        c_gpu,
        size_i * size_j * sizeof(float),
        hipMemcpyDeviceToHost));

    double mse = 0.0;
    double ref_mean_square = 0.0;
    for (int32_t i = 0; i < size_i; ++i) {
        for (int32_t j = 0; j < size_j; ++j) {
            float diff = c_out_host[i * size_j + j] - c[i * size_j + j];
            mse += diff * diff;
            ref_mean_square += c[i * size_j + j] * c[i * size_j + j];
        }
    }
    mse /= size_i * size_j;
    ref_mean_square /= size_i * size_j;
    float rmse = std::sqrt(mse);
    float rel_rmse = rmse / std::sqrt(ref_mean_square);

    if (phase == Phase::BENCHMARK) {
        printf("  %8.02e", rel_rmse);
    }

    if (rel_rmse > 1e-5) {
        if (phase == Phase::BENCHMARK) {
            printf("  %9s  %7s", "-", "-");
        }
    } else {
        double target_time_ms = 200.0;
        double elapsed_ms = benchmark_ms(
            target_time_ms,
            4,
            [&]() {
                if (workspace_size > 0) {
                    CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));
                }
            },
            [&]() {
                Impl::run(size_i, size_j, size_k, a_gpu, b_gpu, c_gpu, workspace_gpu);
            });

        if (phase == Phase::BENCHMARK) {
            double tflop = 2.0 * size_i * size_k * size_j * 1e-12;
            printf("  %9.02f  %7.02f", elapsed_ms, tflop / (elapsed_ms * 1e-3));

            results.elapsed_ms[{size_i, size_j, size_k}] = elapsed_ms;
        }
    }

    printf("\n");

    CUDA_CHECK(hipFree(a_gpu));
    CUDA_CHECK(hipFree(b_gpu));
    CUDA_CHECK(hipFree(c_gpu));
    if (workspace_size > 0) {
        CUDA_CHECK(hipFree(workspace_gpu));
    }
}

template <typename Impl>
BenchmarkResults run_all_configs(
    Phase phase,
    TestData const &data,
    std::vector<BenchmarkConfig> const &configs) {
    auto results = BenchmarkResults{Impl::name};
    if (phase == Phase::WARMUP) {
        printf("warmup %s:\n\n", Impl::name);
    } else {
        printf("%s:\n\n", Impl::name);
        printf(
            "  %-6s  %-6s  %-6s  %-8s  %-9s  %-7s\n",
            "size_i",
            "size_j",
            "size_k",
            "RRMSE",
            "time (ms)",
            "TFLOP/s");
        printf(
            "  %-6s  %-6s  %-6s  %-8s  %-9s  %-7s\n",
            "------",
            "------",
            "------",
            "--------",
            "---------",
            "-------");
    }
    for (auto const &config : configs) {
        run_config<Impl>(phase, data, config, results);
    }
    printf("\n");
    return results;
}

#ifdef HAS_LAB_4_BASELINE_IMPL

struct MatmulL1Reg {
    constexpr static char const *name = "matmul_l1_reg";

    static size_t get_workspace_size(int32_t size_i, int32_t size_j, int32_t size_k) {
        return 0;
    }

    static void
    run(int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c,
        void *workspace) {
        matmul_l1_reg::launch_matmul_l1_reg(size_i, size_j, size_k, a, b, c);
    }
};

#endif

struct MatmulImproved {
    constexpr static char const *name = "matmul_improved";

    static size_t get_workspace_size(int32_t size_i, int32_t size_j, int32_t size_k) {
        return 0;
    }

    static void
    run(int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c,
        void *workspace) {
        matmul_improved::launch_matmul_improved(size_i, size_j, size_k, a, b, c);
    }
};

struct MatmulImprovedReduce {
    constexpr static char const *name = "matmul_improved_reduce";

    static size_t get_workspace_size(int32_t size_i, int32_t size_j, int32_t size_k) {
        return matmul_improved_reduce::get_workspace_size(size_i, size_j, size_k);
    }

    static void
    run(int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c,
        void *workspace) {
        matmul_improved_reduce::launch_matmul_improved_reduce(
            size_i,
            size_j,
            size_k,
            a,
            b,
            c,
            workspace);
    }
};

std::vector<BenchmarkResults> run_all_impls(
    Phase phase,
    TestData const &data,
    std::vector<BenchmarkConfig> const &configs) {
    auto results = std::vector<BenchmarkResults>{};
#ifdef HAS_LAB_4_BASELINE_IMPL
    results.push_back(run_all_configs<MatmulL1Reg>(phase, data, configs));
#endif
    results.push_back(run_all_configs<MatmulImproved>(phase, data, configs));
    results.push_back(run_all_configs<MatmulImprovedReduce>(phase, data, configs));
    return results;
}

void write_json_results(
    std::string const &path,
    std::vector<BenchmarkResults> const &results) {
    auto file = std::ofstream(path);
    file << "{\n";
    for (int32_t i = 0; i < results.size(); ++i) {
        auto const &result = results.at(i);
        file << "  \"" << result.name << "\": [\n";
        int32_t j = 0;
        for (auto const &[config, elapsed_ms] : result.elapsed_ms) {
            auto [size_i, size_j, size_k] = config;
            double tflop = 2.0 * size_i * size_k * size_j * 1e-12;
            double tflop_per_sec = tflop / (elapsed_ms * 1e-3);
            file << "    {\n";
            file << "      \"size_i\": " << size_i << ",\n";
            file << "      \"size_j\": " << size_j << ",\n";
            file << "      \"size_k\": " << size_k << ",\n";
            file << "      \"elapsed_ms\": " << elapsed_ms << ",\n";
            file << "      \"tflop_per_sec\": " << tflop_per_sec << "\n";
            file << "    }";
            if (j + 1 < result.elapsed_ms.size()) {
                file << ",";
            }
            file << "\n";
            ++j;
        }
        file << "  ]";
        if (i + 1 < results.size()) {
            file << ",";
        }
        file << "\n";
    }
    file << "}\n";
}

int main(int argc, char **argv) {
    std::string test_data_dir = ".";
    if (char *c_str_test_data_dir = std::getenv("MATMUL_TEST_DATA_DIR_2")) {
        test_data_dir = c_str_test_data_dir;
    }

    auto configs = std::vector<BenchmarkConfig>{
        {3072, 3072, 3072},
        {512, 3072, 3072},
        {256, 3072, 3072},
        {128, 3072, 3072},
        {64, 3072, 3072},
        {32, 3072, 3072},
        {16, 3072, 3072},
        {1, 3072, 3072},
        {256, 256, 256},
        {256, 256, 1024},
        {256, 256, 8192},
        {128, 128, 32768},
    };
    auto data = read_test_data(test_data_dir, configs);
    run_all_impls(Phase::WARMUP, data, configs);
    auto results = run_all_impls(Phase::BENCHMARK, data, configs);

    for (int32_t j = 1; j < results.size(); ++j) {
        for (int32_t i = j; i > 0;) {
            --i;
            auto const &first = results.at(i);
            auto const &second = results.at(j);
            printf("\nspeedups %s -> %s:\n\n", first.name, second.name);
            printf("  %-6s  %-6s  %-6s  %-7s\n", "size_i", "size_j", "size_k", "speedup");
            printf("  %-6s  %-6s  %-6s  %-7s\n", "------", "------", "------", "-------");
            for (auto const &config : configs) {
                auto size_i = config.size_i;
                auto size_j = config.size_j;
                auto size_k = config.size_k;
                printf("  %6d  %6d  %6d", size_i, size_j, size_k);
                auto it_first = first.elapsed_ms.find({size_i, size_j, size_k});
                auto it_second = second.elapsed_ms.find({size_i, size_j, size_k});
                if (it_first != first.elapsed_ms.end() &&
                    it_second != second.elapsed_ms.end()) {
                    printf("  %6.02fx", it_first->second / it_second->second);
                } else {
                    printf("  %7s", "-");
                }
                printf("\n");
            }
        }
    }

    write_json_results("out/results.json", results);

    return 0;
}